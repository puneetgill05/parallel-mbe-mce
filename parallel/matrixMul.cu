#include "hip/hip_runtime.h"
//
// Created by puneet on 11/05/25.
//

#include "GMBE.h"


#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i] + 0.0f;
}


int main1() {
    int N = 1 << 20; // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize inputs
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy inputs to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel (256 threads per block)
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 100; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << "\n";
    }

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
